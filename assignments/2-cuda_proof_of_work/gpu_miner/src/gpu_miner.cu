#include "hip/hip_runtime.h"
#include <inttypes.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "../include/utils.cuh"

__constant__ BYTE difficulty[SHA256_HASH_SIZE] =
    "0000099999999999999999999999999999999999999999999999999999999999";

// TODO: Implement function to search for all nonces from 1 through MAX_NONCE
// (inclusive) using CUDA Threads
__global__ void findNonce(uint64_t *gpu_nonce, size_t *gpu_current_length,
                          BYTE *gpu_block_hash, BYTE *gpu_block_content) {
  char nonce_string[NONCE_SIZE];
  BYTE gpu_block_content_copy[BLOCK_SIZE];
  d_strcpy((char *)gpu_block_content_copy, (char *)gpu_block_content);
  BYTE gpu_block_hash_copy[BLOCK_SIZE];

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (uint64_t i = index; i <= MAX_NONCE; i += stride) {
    if (gpu_block_hash[0] != 0) break;

    intToString(i, nonce_string);
    d_strcpy((char *)gpu_block_content_copy + *gpu_current_length,
             nonce_string);

    apply_sha256(gpu_block_content_copy,
                 d_strlen((const char *)gpu_block_content_copy),
                 gpu_block_hash_copy, 1);

    if (compare_hashes(gpu_block_hash_copy, difficulty) <= 0) {
      d_strcpy((char *)gpu_block_hash, (char *)gpu_block_hash_copy);
      *gpu_nonce = i;
      break;
    }
  }
}

int main(int argc, char **argv) {
  BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE],
      hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
      tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2],
      hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
      tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE],
      block_content[BLOCK_SIZE];
  BYTE block_hash[SHA256_HASH_SIZE] =
      "0000000000000000000000000000000000000000000000000000000000000000";
  uint64_t nonce = 0;
  uint64_t *gpu_nonce;
  BYTE *gpu_block_content;
  BYTE *gpu_block_hash;
  size_t *gpu_current_length;
  size_t current_length;

  // Top hash
  apply_sha256(tx1, strlen((const char *)tx1), hashed_tx1, 1);
  apply_sha256(tx2, strlen((const char *)tx2), hashed_tx2, 1);
  apply_sha256(tx3, strlen((const char *)tx3), hashed_tx3, 1);
  apply_sha256(tx4, strlen((const char *)tx4), hashed_tx4, 1);
  strcpy((char *)tx12, (const char *)hashed_tx1);
  strcat((char *)tx12, (const char *)hashed_tx2);
  apply_sha256(tx12, strlen((const char *)tx12), hashed_tx12, 1);
  strcpy((char *)tx34, (const char *)hashed_tx3);
  strcat((char *)tx34, (const char *)hashed_tx4);
  apply_sha256(tx34, strlen((const char *)tx34), hashed_tx34, 1);
  strcpy((char *)tx1234, (const char *)hashed_tx12);
  strcat((char *)tx1234, (const char *)hashed_tx34);
  apply_sha256(tx1234, strlen((const char *)tx34), top_hash, 1);

  // prev_block_hash + top_hash
  strcpy((char *)block_content, (const char *)prev_block_hash);
  strcat((char *)block_content, (const char *)top_hash);
  current_length = strlen((char *)block_content);

  hipEvent_t start, stop;
  startTiming(&start, &stop);

  hipMalloc((void **)&gpu_nonce, sizeof(uint64_t));
  hipMalloc((void **)&gpu_block_content, BLOCK_SIZE);
  hipMalloc((void **)&gpu_block_hash, SHA256_HASH_SIZE);
  hipMalloc((void **)&gpu_current_length, sizeof(size_t));

  hipMemcpy((void *)gpu_current_length, (void *)&current_length,
             sizeof(size_t), hipMemcpyHostToDevice);
  hipMemcpy((void *)gpu_nonce, (void *)&nonce, sizeof(uint64_t),
             hipMemcpyHostToDevice);
  hipMemcpy((void *)gpu_block_content, (void *)block_content, BLOCK_SIZE,
             hipMemcpyHostToDevice);
  hipMemset(gpu_block_hash, 0, 1);

  int blockSize = 256;
  int numBlocks = (MAX_NONCE + blockSize - 1) / blockSize;
  findNonce<<<numBlocks, blockSize>>>(gpu_nonce, gpu_current_length,
                                      gpu_block_hash, gpu_block_content);

  hipDeviceSynchronize();

  hipMemcpy((void *)&nonce, (void *)gpu_nonce, sizeof(uint64_t),
             hipMemcpyDeviceToHost);
  hipMemcpy((void *)block_hash, (void *)gpu_block_hash, SHA256_HASH_SIZE,
             hipMemcpyDeviceToHost);

  hipFree(gpu_nonce);
  hipFree(gpu_block_content);
  hipFree(gpu_block_hash);
  hipFree(gpu_current_length);
  float seconds = stopTiming(&start, &stop);
  printResult(block_hash, nonce, seconds);

  return 0;
}
